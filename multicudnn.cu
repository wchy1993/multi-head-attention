#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <hipblas.h>
#include <assert.h>
#include <math.h>

const int d_model = 512;
const int num_heads = 16;
const int head_dim = d_model / num_heads;
const int batch_size = 8;
const int seq_len = 64;

void check_cudnn_error(cudnnStatus_t status) {
    if (status != CUDNN_STATUS_SUCCESS) {
        fprintf(stderr, "Error on line %d: %s\n", __LINE__, cudnnGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}

void check_cublas_error(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "Error on line %d: %d\n", __LINE__, status);
        exit(EXIT_FAILURE);
    }
}

void check_cuda_error(hipError_t error) {
    if (error != hipSuccess) {
        fprintf(stderr, "Error on line %d: %s\n", __LINE__, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void multihead_attention(cudnnHandle_t cudnn, hipblasHandle_t cublas, float *query, float *key, float *value, float *output, float *WQ, float *WK, float *WV, float *WO) {
    float *Q, *K, *V, *scores;
    size_t tensor_size = batch_size * seq_len * d_model * sizeof(float);
    hipMalloc((void **)&Q, tensor_size);
    hipMalloc((void **)&K, tensor_size);
    hipMalloc((void **)&V, tensor_size);
    hipMalloc((void **)&scores, batch_size * num_heads * seq_len * seq_len * sizeof(float));
    hipMalloc((void **)&Q, tensor_size);
    hipMalloc((void **)&K, tensor_size);
    hipMalloc((void **)&V, tensor_size);
    hipMalloc((void **)&scores, batch_size * num_heads * seq_len * seq_len * sizeof(float));

    float alpha = 1.0f, beta = 0.0f;
    for (int i = 0; i < num_heads; i++) {
        int offset = i * head_dim;
        hipblasSgemm(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, seq_len, batch_size, d_model, &alpha, query, seq_len, WQ + offset, d_model, &beta, Q + offset, seq_len);
        hipblasSgemm(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, seq_len, batch_size, d_model, &alpha, key, seq_len, WK + offset, d_model, &beta, K + offset, seq_len);
        hipblasSgemm(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, seq_len, batch_size, d_model, &alpha, value, seq_len, WV + offset, d_model, &beta, V + offset, seq_len);
    }

    hipblasSgemm(cublas, HIPBLAS_OP_N, HIPBLAS_OP_T, seq_len, seq_len, d_model, &alpha, Q, seq_len, K, seq_len, &beta, scores, seq_len);

    cudnnTensorDescriptor_t scores_desc;
    cudnnCreateTensorDescriptor(&scores_desc);
    cudnnSetTensor4dDescriptor(scores_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, batch_size * num_heads, 1, seq_len, seq_len);
    cudnnSoftmaxForward(cudnn, CUDNN_SOFTMAX_FAST, CUDNN_SOFTMAX_MODE_INSTANCE, &alpha, scores_desc, scores, &beta, scores_desc, scores);

    float *context;
    hipMalloc((void **)&context, tensor_size);

    hipblasSgemm(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, seq_len, batch_size, seq_len, &alpha, scores, seq_len, V, seq_len, &beta, context, seq_len);

    hipblasSgemm(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, seq_len, batch_size, d_model, &alpha, context, seq_len, WO, d_model, &beta, output, seq_len);

    hipFree(Q);
    hipFree(K);
    hipFree(V);
    hipFree(scores);
    hipFree(context);
    cudnnDestroyTensorDescriptor(scores_desc);
}


int main() {
    cudnnHandle_t cudnn;
    check_cudnn_error(cudnnCreate(&cudnn));

    hipblasHandle_t cublas;
    check_cublas_error(hipblasCreate(&cublas));
    size_t tensor_size = batch_size * seq_len * d_model * sizeof(float);
    float *query, *key, *value, *output;
    hipMalloc((void **)&query, tensor_size);
    hipMalloc((void **)&key, tensor_size);
    hipMalloc((void **)&value, tensor_size);
    hipMalloc((void **)&output, tensor_size);

    // Initialize the input tensors (query, key, and value) with appropriate values

    size_t weights_size = d_model * d_model * sizeof(float);
    float *WQ, *WK, *WV, *WO;
    hipMalloc((void **)&WQ, weights_size);
    hipMalloc((void **)&WK, weights_size);
    hipMalloc((void **)&WV, weights_size);
    hipMalloc((void **)&WO, weights_size);

    // Initialize the weight matrices with appropriate values
 

    // Create CUDA events for timing
    hipEvent_t start, stop;
    check_cuda_error(hipEventCreate(&start));
    check_cuda_error(hipEventCreate(&stop));

    // Record the start event
    check_cuda_error(hipEventRecord(start, 0));

    multihead_attention(cudnn, cublas, query, key, value, output, WQ, WK, WV, WO);

    // Record the stop event
    check_cuda_error(hipEventRecord(stop, 0));
    check_cuda_error(hipEventSynchronize(stop));

    // Calculate elapsed time in milliseconds
    float elapsedTime;
    check_cuda_error(hipEventElapsedTime(&elapsedTime, start, stop));

    printf("Time elapsed for multihead_attention: %f ms\n", elapsedTime);
    // Process the output tensor as needed (e.g., print, save to a file, etc.)
 

    hipFree(query);
    hipFree(key);
    hipFree(value);
    hipFree(output);
    hipFree(WQ);
    hipFree(WK);
    hipFree(WV);
    hipFree(WO);

    // Destroy CUDA events
    check_cuda_error(hipEventDestroy(start));
    check_cuda_error(hipEventDestroy(stop));

    cudnnDestroy(cudnn);
    hipblasDestroy(cublas);

    return 0;
}

        
